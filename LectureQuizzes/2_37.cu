#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define ARRAY_SIZE  100
bool atomic = true;

#define BLOCK_WIDTH 1000

void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++)  { printf("%d ", array[i]); }
    printf("}\n");
}

__global__ void increment_naive(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	g[i] = g[i] + 1;
}

__global__ void increment_atomic(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	atomicAdd(& g[i], 1);
}

int main(int argc,char **argv)
{   
    GpuTimer timer;
    if (atomic) {
        printf("atomic %d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);
    } else {
        printf("%d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);
    }

    // declare and allocate host memory
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
 
    // declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    // launch the kernel - comment out one of these
    timer.Start();
    
    // Instructions: This program is needed for the next quiz
    // uncomment increment_naive to measure speed and accuracy 
    // of non-atomic increments or uncomment increment_atomic to
    // measure speed and accuracy of  atomic icrements
    if (atomic) {
        increment_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    } else {
        increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    }
    timer.Stop();
    
    // copy back the array of sums from GPU and print
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    // print_array(h_array, ARRAY_SIZE);
    printf("Time elapsed = %g ms\n", timer.Elapsed());
 
    // free GPU memory allocation and exit
    hipFree(d_array);
    return 0;
}