
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
    printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}


int main(int argc,char **argv)
{
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("That's all!\n");

    return 0;
}